#include "hip/hip_runtime.h"
/**
 * @file work_oriented.cu
 * @author Muhammad Osama (mosama@ucdavis.edu)
 * @brief Sparse Matrix-Vector Multiplication example.
 * @version 0.1
 * @date 2022-02-03
 *
 * @copyright Copyright (c) 2022
 *
 */

#include "spmv.hxx"

using namespace loops;

template <std::size_t threads_per_block,
          typename index_t,
          typename offset_t,
          typename type_t>
__global__ void __launch_bounds__(threads_per_block, 2)
    merge_spmv(std::size_t rows,
               std::size_t cols,
               std::size_t nnz,
               offset_t* offsets,
               index_t* indices,
               const type_t* values,
               const type_t* x,
               type_t* y) {
  using setup_t =
      schedule::setup<schedule::algorithms_t::work_oriented, threads_per_block,
                      1, index_t, offset_t, std::size_t, std::size_t>;

  setup_t config(offsets, rows, nnz);
  auto map = config.init();

  /// Accumulate the complete tiles.
  type_t sum = 0;
  for (auto row : config.tiles(map)) {
    for (auto nz : config.atoms(row, map)) {
      sum += values[nz] * x[indices[nz]];
    }
    y[row] = sum;
    sum = 0;
  }

  // Interesting use of syncthreads to ensure all remaining tiles get processed
  // at the same time, possibly causing less thread divergence among the threads
  // in the same warp.
  __syncthreads();

  /// Process remaining tiles.
  for (auto row : config.remainder_tiles(map)) {
    for (auto nz : config.remainder_atoms(map)) {
      sum += values[nz] * x[indices[nz]];
    }
    /// Accumulate the remainder.
    if (sum != 0)
      atomicAdd(&(y[row]), sum);
  }
}

int main(int argc, char** argv) {
  using index_t = int;
  using offset_t = int;
  using type_t = float;

  // ... I/O parameters, mtx, etc.
  parameters_t parameters(argc, argv);

  csr_t<index_t, offset_t, type_t> csr;
  matrix_market_t<index_t, offset_t, type_t> mtx;
  csr.from_coo(mtx.load(parameters.filename));

  // Input and output vectors.
  vector_t<type_t> x(csr.rows);
  vector_t<type_t> y(csr.rows);

  // Generate random numbers between [0, 1].
  generate::random::uniform_distribution(x.begin(), x.end(), 1, 10);

  // Create a schedule.
  constexpr std::size_t block_size = 128;

  /// Set-up kernel launch parameters and run the kernel.
  hipStream_t stream;
  hipStreamCreate(&stream);

  /// Launch 2 x (SM Count) number of blocks.
  /// Weirdly enough, a really high number here might cause it to fail.
  loops::device::properties_t props;
  std::size_t grid_size = 2 * props.multi_processor_count();

  launch::non_cooperative(
      stream, merge_spmv<block_size, index_t, offset_t, type_t>, grid_size,
      block_size, csr.rows, csr.cols, csr.nnzs, csr.offsets.data().get(),
      csr.indices.data().get(), csr.values.data().get(), x.data().get(),
      y.data().get());

  hipStreamSynchronize(stream);

  /// Validation code, can be safely ignored.
  if (parameters.validate) {
    auto h_y = cpu::spmv(csr, x);

    std::size_t errors = util::equal(
        y.data().get(), h_y.data(), csr.rows,
        [](const type_t a, const type_t b) { return std::abs(a - b) > 1e-2; },
        parameters.verbose);

    std::cout << "Matrix:\t\t" << extract_filename(parameters.filename)
              << std::endl;
    std::cout << "Dimensions:\t" << csr.rows << " x " << csr.cols << " ("
              << csr.nnzs << ")" << std::endl;
    std::cout << "Errors:\t\t" << errors << std::endl;
  }
}