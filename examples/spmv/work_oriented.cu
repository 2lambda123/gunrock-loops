#include "hip/hip_runtime.h"
/**
 * @file spmv.cu
 * @author Muhammad Osama (mosama@ucdavis.edu)
 * @brief Sparse Matrix-Vector Multiplication example.
 * @version 0.1
 * @date 2022-02-03
 *
 * @copyright Copyright (c) 2022
 *
 */

#include "spmv.hxx"

using namespace loops;

template <std::size_t threads_per_block,
          typename index_t,
          typename offset_t,
          typename type_t>
__global__ void __launch_bounds__(threads_per_block, 2)
    merge_spmv(std::size_t rows,
               std::size_t cols,
               std::size_t nnz,
               offset_t* offsets,
               index_t* indices,
               const type_t* values,
               const type_t* x,
               type_t* y) {
  using setup_t = schedule::setup<schedule::algorithms_t::work_oriented,
                                  threads_per_block, 1, index_t, offset_t>;

  setup_t config(offsets, rows, nnz);
  auto map = config.init();

  /// Accumulate the complete tiles.
  type_t sum = 0;
  for (auto row : config.tiles(map)) {
    for (auto nz : config.atoms(row, map)) {
      sum += values[nz] * x[indices[nz]];
    }
    y[row] = sum;
    sum = 0;
  }

  int remainder_row = map.second.first;
  for (auto nz : config.remainder_atoms(map)) {
    sum += values[nz] * x[indices[nz]];
  }

  /// Accumulate the remainder.
  if (sum != 0)
    atomicAdd(&(y[remainder_row]), sum);
}

int main(int argc, char** argv) {
  using index_t = int;
  using offset_t = int;
  using type_t = float;

  // ... I/O parameters, mtx, etc.
  parameters_t parameters(argc, argv);

  csr_t<index_t, offset_t, type_t> csr;
  matrix_market_t<index_t, offset_t, type_t> mtx;
  csr.from_coo(mtx.load(parameters.filename));

  // Input and output vectors.
  vector_t<type_t> x(csr.rows);
  vector_t<type_t> y(csr.rows);

  std::cout << "# Rows: " << csr.rows << std::endl;
  std::cout << "# Columns: " << csr.cols << std::endl;

  // Generate random numbers between [0, 1].
  generate::random::uniform_distribution(x.begin(), x.end(), 1, 10);

  // Create a schedule.
  constexpr std::size_t block_size = 128;

  /// Set-up kernel launch parameters and run the kernel.
  hipStream_t stream;
  hipStreamCreate(&stream);

  std::size_t grid_size =
      (((csr.rows + csr.nnzs) + block_size) - 1) / block_size;
  launch::non_cooperative(
      stream, merge_spmv<block_size, index_t, offset_t, type_t>, grid_size,
      block_size, csr.rows, csr.cols, csr.nnzs, csr.offsets.data().get(),
      csr.indices.data().get(), csr.values.data().get(), x.data().get(),
      y.data().get());

  hipStreamSynchronize(stream);

  /// Validation code, can be safely ignored.
  if (parameters.validate) {
    auto h_y = cpu::spmv(csr, x);

    std::size_t errors = util::equal(
        y.data().get(), h_y.data(), csr.rows,
        [](const type_t a, const type_t b) { return std::abs(a - b) > 1e-2; },
        parameters.verbose);

    std::cout << "Matrix:\t\t" << extract_filename(parameters.filename)
              << std::endl;
    std::cout << "Dimensions:\t" << csr.rows << " x " << csr.cols << " ("
              << csr.nnzs << ")" << std::endl;
    std::cout << "Errors:\t\t" << errors << std::endl;
  }
}